#include "hip/hip_runtime.h"
/***************************************************************************
 * Copyright 2025 The SpInfer Authors. All rights reserved.
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * http://www.apache.org/licenses/LICENSE-2.0
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 ***************************************************************************/
#include "./MatMulUtilities.cuh"
#include "./Reduction_Kernel.cuh"
#include "./SpMM_Kernel.cuh"
// #include "./SpMV_Kernel.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

template<typename TilingConfig>
static void SpMM_SplitK_Kernel_Ex_bitmap_v3(hipStream_t stream,
                                  const half*  A,
                                  const half* Compressed_A,
                                  const int*   TileOffsets,
                                  const int* TileOffsets_Median,
                                  const uint64_t*   bitmap,
                                  const int* max_nnz_intile,
                                  const half*  B,
                                  half*        Reduction_Workspace,
                                  const int    M_Global,
                                  const int    N_Global,
                                  const int    K_Global,
                                  int          Split_K)
{
    // Maximum size of dynamic SMEM = size between B tile double buffer & C shared memory
    // ? Dynamic shared memory ? : Can be determined with kernel runtime
    // 13b: 2304
    static int SHMEM_SZ = max((TilingConfig::TILE_N * TILE_K) * sizeof(half) * 2 + 2304 * sizeof(half) + (TilingConfig::TILE_BITMAP_M_V3 * TilingConfig::TILE_BITMAP_K_V3) * sizeof(uint64_t),
                              (TilingConfig::TILE_M + PADDING_SHARED_MEM_FOR_C) * TilingConfig::TILE_N * sizeof(float));
    // ### hipFuncSetAttribute : change attribute
    // To change dynamic shared memory with 48KB over generally.
    // Inside : changes the runtime metadata of the kernel
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        SpMM_Kernel_bitmap_v3<TilingConfig>), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ);
    int dimN =
        max(N_Global / TilingConfig::TILE_N, 1);  // max(N_Global/TilingConfig::TILE_N,1) used when N=8, TILE_N=16
    int  dimM = M_Global * Split_K / TilingConfig::TILE_M;
    dim3 GridDim(dimN, dimM, 1);  // Grid Size is increased due to SplitK for higher SM occupancy
    dim3 BlockDim(WARP_SIZE * TilingConfig::BLOCK_WARPS, 1, 1);
    SpMM_Kernel_bitmap_v3<TilingConfig><<<GridDim, BlockDim, SHMEM_SZ, stream>>>(
        A, Compressed_A, TileOffsets, TileOffsets_Median, bitmap, max_nnz_intile, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);
}

template<typename TilingConfig>
static void SpMV_SplitK_Kernel_Ex_bitmap_v3(hipStream_t stream,
                                  const half*  A,
                                  const half* Compressed_A,
                                  const int*   TileOffsets,
                                  const int* TileOffsets_Median,z                       
                                  const uint64_t*   bitmap,
                                  const int* max_nnz_intile,
                                  const half*  B,
                                  half*        Reduction_Workspace,
                                  const int    M_Global,
                                  const int    N_Global,
                                  const int    K_Global,
                                  int          Split_K)
{
    // Maximum size of dynamic SMEM = size between B tile double buffer & C shared memory
    // ? Dynamic shared memory ? : Can be determined with kernel runtime
    // 13b: 2304
    static int SHMEM_SZ = max((TilingConfig::TILE_N * TILE_K) * sizeof(half) * 2 + 2304 * sizeof(half) + (TilingConfig::TILE_BITMAP_M_V3 * TilingConfig::TILE_BITMAP_K_V3) * sizeof(uint64_t),
                              (TilingConfig::TILE_M + PADDING_SHARED_MEM_FOR_C) * TilingConfig::TILE_N * sizeof(float));
    // ### hipFuncSetAttribute : change attribute
    // To change dynamic shared memory with 48KB over generally.
    // Inside : changes the runtime metadata of the kernel
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        SpMV_Kernel_bitmap_v3<TilingConfig>), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ);
    int dimN =
        max(N_Global / TilingConfig::TILE_N, 1);  // max(N_Global/TilingConfig::TILE_N,1) used when N=8, TILE_N=16
    int  dimM = M_Global * Split_K / TilingConfig::TILE_M;
    dim3 GridDim(dimN, dimM, 1);  // Grid Size is increased due to SplitK for higher SM occupancy
    dim3 BlockDim(WARP_SIZE * TilingConfig::BLOCK_WARPS, 1, 1);
    SpMV_Kernel_bitmap_v3<TilingConfig><<<GridDim, BlockDim, SHMEM_SZ, stream>>>(
        A, Compressed_A, TileOffsets, TileOffsets_Median, bitmap, max_nnz_intile, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);
}


hipError_t SpMV_SplitK_API_bitmap_v3(hipStream_t stream,
                            const half*  A,
                            const half*  Compressed_A,
                            const int*   TileOffsets,
                            const int* TileOffsets_Median,
                            const uint64_t* bitmap,
                            const int* max_nnz_intile,
                            const half*  B,
                            half*        C,
                            const int    M_Global,
                            const int    N_Global,
                            const int    K_Global,
                            half*        Reduction_Workspace,  // Identical workspace for all SpMM kernel launchesSpMM_SplitK_Kernel_Ex_bitmap
                            int          Split_K)
{
    // ### hipStream_t : To control Asynchronous task execution(especially for hipMemcpyAsync)
    half* SpMM_SplitK_OutputPTR;
    if (Split_K == 1)
        SpMM_SplitK_OutputPTR = C;
    else
        SpMM_SplitK_OutputPTR = Reduction_Workspace;
    // Batched SpMM
    switch (N_Global) {
        case 1:
            // ### template<int BLOCK_ROW_WARPS_, int BLOCK_COL_WARPS_, int WARP_COL_TENSORS_, int N8_ = 0>
            // struct TilingConfigBitmapV3
            // - BLOCK_ROW_WARPS_ : Number of warps to handle TILE_M
            // - BLOCK_COL_WARPS_ : Number of warps to handle TILE_N(always 1)
            // - WARP_COL_TENSORS_ : ???
            // - N8_ : only for N_Global=8
            SpMV_SplitK_Kernel_Ex_bitmap_v3<TilingConfigBitmapV3_N1<4, 1, 1, 1>>(
                stream, A, Compressed_A, TileOffsets, TileOffsets_Median, bitmap, max_nnz_intile, B, SpMM_SplitK_OutputPTR, M_Global, N_Global, K_Global, Split_K);
            break;
        default:
            printf("MV_Sparse_API Error: Only vector supported %d!\n", N_Global);
            return hipErrorUnknown;
    }
    
    //
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess)
        return Error;

    if (Split_K == 1)
        return Error;
    
    dim3 GridDim((M_Global * N_Global) / 256, 1, 1);
    dim3 BlockDim(WARP_SIZE, 1, 1);
    SplitK_Reduction<<<GridDim, BlockDim, 0, stream>>>(C, Reduction_Workspace, M_Global, N_Global, Split_K);
    return hipGetLastError();
}


hipError_t SpMM_SplitK_API_bitmap_v3(hipStream_t stream,
                            const half*  A,
                            const half*  Compressed_A,
                            const int*   TileOffsets,
                            const int* TileOffsets_Median,
                            const uint64_t* bitmap,
                            const int* max_nnz_intile,
                            const half*  B,
                            half*        C,
                            const int    M_Global,
                            const int    N_Global,
                            const int    K_Global,
                            half*        Reduction_Workspace,  // Identical workspace for all SpMM kernel launchesSpMM_SplitK_Kernel_Ex_bitmap
                            int          Split_K)
{
    // ### hipStream_t : To control Asynchronous task execution(especially for hipMemcpyAsync)
    half* SpMM_SplitK_OutputPTR;
    if (Split_K == 1)
        SpMM_SplitK_OutputPTR = C;
    else
        SpMM_SplitK_OutputPTR = Reduction_Workspace;
    // Batched SpMM
    switch (N_Global) {
        case 8:
            // ### template<int BLOCK_ROW_WARPS_, int BLOCK_COL_WARPS_, int WARP_COL_TENSORS_, int N8_ = 0>
            // struct TilingConfigBitmapV3
            // - BLOCK_ROW_WARPS_ : Number of warps to handle TILE_M
            // - BLOCK_COL_WARPS_ : Number of warps to handle TILE_N(always 1)
            // - WARP_COL_TENSORS_ : ???
            // - N8_ : only for N_Global=8
            SpMM_SplitK_Kernel_Ex_bitmap_v3<TilingConfigBitmapV3<4, 1, 1, 1>>(
                stream, A, Compressed_A, TileOffsets, TileOffsets_Median, bitmap, max_nnz_intile, B, SpMM_SplitK_OutputPTR, M_Global, N_Global, K_Global, Split_K);
            break;
        case 16:
            SpMM_SplitK_Kernel_Ex_bitmap_v3<TilingConfigBitmapV3<4, 1, 1>>(
                stream, A, Compressed_A, TileOffsets, TileOffsets_Median, bitmap, max_nnz_intile, B, SpMM_SplitK_OutputPTR, M_Global, N_Global, K_Global, Split_K);
            break;
        case 32:
            SpMM_SplitK_Kernel_Ex_bitmap_v3<TilingConfigBitmapV3<4, 1, 2>>(
                stream, A, Compressed_A, TileOffsets, TileOffsets_Median, bitmap, max_nnz_intile, B, SpMM_SplitK_OutputPTR, M_Global, N_Global, K_Global, Split_K);
            break;
        case 64:
            SpMM_SplitK_Kernel_Ex_bitmap_v3<TilingConfigBitmapV3<4, 1, 4>>(
                stream, A, Compressed_A, TileOffsets, TileOffsets_Median, bitmap, max_nnz_intile, B, SpMM_SplitK_OutputPTR, M_Global, N_Global, K_Global, Split_K);
            break;
        case 128:
            SpMM_SplitK_Kernel_Ex_bitmap_v3<TilingConfigBitmapV3<4, 1, 4>>(
                stream, A, Compressed_A, TileOffsets, TileOffsets_Median, bitmap,  max_nnz_intile, B, SpMM_SplitK_OutputPTR, M_Global, N_Global, K_Global, Split_K);
            break;
        default:
            if (N_Global % 128 == 0)
            SpMM_SplitK_Kernel_Ex_bitmap_v3<TilingConfigBitmapV3<4, 1, 4>>(stream,
                                                                                     A,
                                                                                     Compressed_A,
                                                                                     TileOffsets,
                                                                                     TileOffsets_Median,
                                                                                     bitmap,
                                                                                     max_nnz_intile,
                                                                                     B,
                                                                                     SpMM_SplitK_OutputPTR,
                                                                                     M_Global,
                                                                                     N_Global,
                                                                                     K_Global,
                                                                                     Split_K);
            else {
                printf("MM_Sparse_API Error: Unsupported N dimension %d!\n", N_Global);
                return hipErrorUnknown;
            }
            break;
    }
    
    //
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess)
        return Error;

    if (Split_K == 1)
        return Error;
    
    dim3 GridDim((M_Global * N_Global) / 256, 1, 1);
    dim3 BlockDim(WARP_SIZE, 1, 1);
    SplitK_Reduction<<<GridDim, BlockDim, 0, stream>>>(C, Reduction_Workspace, M_Global, N_Global, Split_K);
    return hipGetLastError();
}
__host__ int InitSparseMatrixA_bitmap(
    half* A_h,
    int M,
    int K,
    int tile_M,  // 8
    int tile_M_median,  // 16
    int tile_M_global,  // 64
    int tile_K,  // 8
    int tile_K_median,  // 64
    int tile_K_global,  // 64
    half** Compressed_Val,
    int** TileOffsets,
    int** TileOffsets_median,
    int** TileOffsets_global,
    uint64_t** bitmap,
    int& max_nnz_count)
{
    // Calculate the number of tiles for each layer
    int num_tiles_M = M / tile_M;
    int num_tiles_K = K / tile_K;
    int num_tiles = num_tiles_M * num_tiles_K;
    
    int num_median_tiles_M = M / tile_M_median;
    int num_median_tiles_K = K / tile_K_median;
    int num_median_tiles = num_median_tiles_M * num_median_tiles_K;

    int num_global_tiles_M = M / tile_M_global;
    int num_global_tiles_K = K / tile_K_global;
    int num_global_tiles = num_global_tiles_M * num_global_tiles_K;

    // Allocate memory for each data structure
    *Compressed_Val = (half*)malloc(M * K * sizeof(half));
    *TileOffsets = (int*)malloc(num_tiles * sizeof(int));
    *TileOffsets_median = (int*)malloc(num_median_tiles * (tile_M_median / tile_M * tile_K_median / tile_K) * sizeof(int));
    *TileOffsets_global = (int*)malloc((num_global_tiles + 1) * sizeof(int));
    *bitmap = (uint64_t*)malloc(num_tiles * sizeof(uint64_t));

    if (*Compressed_Val == nullptr || *TileOffsets == nullptr || 
        *TileOffsets_median == nullptr || *TileOffsets_global == nullptr || *bitmap == nullptr) {
        return -1;
    }

    int val_count = 0;
    int tile_idx = 0;
    int median_offset_idx = 0;
    std::vector<int> global_val_counts(num_global_tiles + 1, 0);
    max_nnz_count = 0;

    // Traverse all global tiles
    for (int global_tile_m = 0; global_tile_m < num_global_tiles_M; ++global_tile_m) {
        for (int global_tile_k = 0; global_tile_k < num_global_tiles_K; ++global_tile_k) {
            int global_row_start = global_tile_m * tile_M_global;
            int global_col_start = global_tile_k * tile_K_global;
            int global_val_count = 0;
            
            int median_val_count = 0;
            (*TileOffsets_median)[median_offset_idx++] = 0;  // The starting offset of each median tile is 0
            // Traverse the median tiles within the global tile (in row order)
            for (int median_tile_m = 0; median_tile_m < tile_M_global / tile_M_median; ++median_tile_m) {
                for (int median_tile_k = 0; median_tile_k < tile_K_global / tile_K_median; ++median_tile_k) {
                    int median_row_start = global_row_start + median_tile_m * tile_M_median;
                    int median_col_start = global_col_start + median_tile_k * tile_K_median;
                    // Process the 2x2 small tile groups within the median tile
                    for (int local_tile_m_group = 0; local_tile_m_group < tile_M_median / tile_M; local_tile_m_group += 2) {
                        for (int local_tile_k_group = 0; local_tile_k_group < tile_K_median / tile_K; local_tile_k_group += 2) {
                            // Process the 2x2 small tile groups in column-major order
                            for (int j = 0; j < 2; ++j) {
                                for (int i = 0; i < 2; ++i) {
                                    int local_tile_k = local_tile_k_group + j;
                                    int local_tile_m = local_tile_m_group + i;

                                    int col_start = median_col_start + local_tile_k * tile_K;
                                    int row_start = median_row_start + local_tile_m * tile_M;

                                    uint64_t tile_bitmap = 0;
                                    int local_val_count = 0;

                                    // Process all elements in the small tile
                                    for (int row_offset = 0; row_offset < tile_M; ++row_offset) {
                                        for (int col_offset = 0; col_offset < tile_K; ++col_offset) {
                                            int row = row_start + row_offset;
                                            int col = col_start + col_offset;

                                            if (row < M && col < K) {
                                                half val = A_h[row * K + col];
                                                if (__half2float(val) != 0.0f) {
                                                    tile_bitmap |= (1ULL << (row_offset * tile_K + col_offset));
                                                    (*Compressed_Val)[val_count++] = val;
                                                    local_val_count++;
                                                    median_val_count++;
                                                    global_val_count++;
                                                }
                                            }
                                        }
                                    }

                                    (*bitmap)[tile_idx] = tile_bitmap;
                                    (*TileOffsets)[tile_idx] = local_val_count;
                                    ++tile_idx;
                                }
                            }
                        }
                    }
                    if(median_tile_m < (tile_M_global / tile_M_median - 1) or median_tile_k < (tile_K_global / tile_K_median - 1)){
                        // Update TileOffsets_median
                        (*TileOffsets_median)[median_offset_idx] = median_val_count;
                        median_offset_idx++;
                    } 

                }
            }

            // Additional padding for global tiles (if necessary)
            int global_padding = (8 - (global_val_count % 8)) % 8;
            for (int p = 0; p < global_padding; ++p) {
                (*Compressed_Val)[val_count++] = __float2half(0.0f);
            }
            global_val_count += global_padding;

            // Update global_val_counts and max_nnz_count
            global_val_counts[global_tile_m * num_global_tiles_K + global_tile_k + 1] = global_val_count;
            if (global_val_count > max_nnz_count) {
                max_nnz_count = global_val_count;
            }
        }
    }

    // Calculate offsets for global tiles
    (*TileOffsets_global)[0] = 0;
    for (int i = 1; i <= num_global_tiles; ++i) {
        global_val_counts[i] += global_val_counts[i - 1];
        (*TileOffsets_global)[i] = global_val_counts[i];
    }

    // Reduce the size of Compressed_Val to the actually required size
    *Compressed_Val = (half*)realloc(*Compressed_Val, val_count * sizeof(half));

    return num_global_tiles;
}

extern "C" void Our_GenSparseMatrixBinFile(char* DenseMatrixFileName,
                                       int   M,
                                       int   K,
                                       char* Compressed_ValFileName,
                                       char* bitmap_TileOffsets_globalFileName,
                                       char* bitmap_TileOffsets_medianFileName,
                                       char* bitmapFileName,
                                       char* max_nnz_intileFileName,
                                       char* OutputSizesFileName)
{
    std::vector<half> host_array(M * K);
    std::ifstream     in(DenseMatrixFileName, std::ios::in | std::ios::binary);
    if (!in.is_open()) {
        printf("file %s cannot be opened, loadDataArrayFromBin fails. \n", DenseMatrixFileName);
        exit(-1);
    }
    size_t loaded_data_size = sizeof(half) * M * K;
    in.seekg(0, in.end);
    in.seekg(0, in.beg);
#ifdef DEBUG_MODE
    printf("Read %ld bytes from %s.\n", loaded_data_size, DenseMatrixFileName);
#endif
    in.read((char*)host_array.data(), loaded_data_size);
    size_t in_get_size = in.gcount();
    if (in_get_size != loaded_data_size) {
        printf("file %s only has %ld, but request %ld, loading DenseMatrix fails! \n",
               DenseMatrixFileName,
               in_get_size,
               loaded_data_size);
        exit(-1);
    }
    in.close();
    // Step 2: Dense to Sparse Transformation
    // Define output pointer
    half* Compressed_Val_cpu_v3 = nullptr;
    int* bitmap_TileOffsets_cpu_v3 = nullptr;
    int* bitmap_TileOffsets_median_cpu_v3 = nullptr;
    int* bitmap_TileOffsets_global_cpu_v3 = nullptr;
    uint64_t* bitmap_cpu_v3 = nullptr;
    int max_nnz_intilev3 = 0;
    // Call InitSparseMatrixA_bitmap
    auto num_gtilesv3 = InitSparseMatrixA_bitmap(host_array.data(), M, K, 8, 16, 64, 8, 64, 64, &Compressed_Val_cpu_v3, &bitmap_TileOffsets_cpu_v3, &bitmap_TileOffsets_median_cpu_v3, &bitmap_TileOffsets_global_cpu_v3, &bitmap_cpu_v3, max_nnz_intilev3);
    auto local_tile_numv3 = 8*8;
    auto median_tile_numv3 = 4*1;
    auto num_ltilesv3 = num_gtilesv3*local_tile_numv3;
    auto num_mtilesv3 = num_gtilesv3*median_tile_numv3;
    int val_count_v3 = bitmap_TileOffsets_global_cpu_v3[num_gtilesv3]; // The offset of the last tile is the total number of non - zero values after compression
    // Adjust max_nnz_intilev3 to a multiple of 64
    if (max_nnz_intilev3 % 64 != 0) {
        max_nnz_intilev3 = ((max_nnz_intilev3 / 64) + 1) * 64;
    }
    printf("num_global_tiles: %d, bitmap v3 NNZ: %d, max_nnz_intilev3: %d \n", num_gtilesv3, val_count_v3, max_nnz_intilev3);
    // Step 3: Write to FILE(OutputSizesFileName), size[4]
    //         Write to FILE(Compressed_ValFileName), size[val_count_v3]
    //         Write to FILE(bitmap_TileOffsets_globalFileName), size[num_gtilesv3 + 1], FILE(bitmap_TileOffsets_medianFileName), size[num_mtilesv3], FILE(BitmapFileName), size[num_ltilesv3]
    //         Write to FILE(max_nnz_intileFileName), size[1]
    std::ofstream out_SizesFile(OutputSizesFileName, std::ios::out | std::ios::binary);   // 4
    std::ofstream out_CompressedvalFile(Compressed_ValFileName, std::ios::out | std::ios::binary); // val_count_v3
    std::ofstream out_BitmapglobalFile(bitmap_TileOffsets_globalFileName, std::ios::out | std::ios::binary);  // num_gtilesv3 + 1
    std::ofstream out_BitmapmedianFile(bitmap_TileOffsets_medianFileName, std::ios::out | std::ios::binary);  // num_mtilesv3
    std::ofstream out_BitmapFile(bitmapFileName, std::ios::out | std::ios::binary);       // num_ltilesv3
    std::ofstream out_NnzintileFile(max_nnz_intileFileName, std::ios::out | std::ios::binary);     // 1
    if (!out_SizesFile.is_open() || !out_CompressedvalFile.is_open() || !out_BitmapglobalFile.is_open() || !out_BitmapmedianFile.is_open() || !out_BitmapFile.is_open() | !out_NnzintileFile.is_open()) {
        printf("Our_GenSparseMatrixBinFile() ERROR: file %s, %s, %s, %s, %s or %s cannot be opened or creaetd. \n",
               OutputSizesFileName, Compressed_ValFileName,
               bitmap_TileOffsets_globalFileName, bitmap_TileOffsets_medianFileName,
               bitmapFileName, max_nnz_intileFileName);
        exit(-1);
    }
    out_SizesFile.write((char*)&val_count_v3, sizeof(int));
    num_gtilesv3++;
    out_SizesFile.write((char*)&num_gtilesv3, sizeof(int));
    out_SizesFile.write((char*)&num_mtilesv3, sizeof(int));
    out_SizesFile.write((char*)&num_ltilesv3, sizeof(int));
    out_SizesFile.close();
    out_CompressedvalFile.write((char*)Compressed_Val_cpu_v3, sizeof(half) * val_count_v3);
    out_CompressedvalFile.close();
    out_BitmapglobalFile.write((char*)bitmap_TileOffsets_global_cpu_v3, sizeof(int) * num_gtilesv3);
    out_BitmapglobalFile.close();
    out_BitmapmedianFile.write((char*)bitmap_TileOffsets_median_cpu_v3, sizeof(int) * num_mtilesv3);
    out_BitmapmedianFile.close();
    out_BitmapFile.write((char*)bitmap_cpu_v3, sizeof(uint64_t) * num_ltilesv3);
    out_BitmapFile.close();
    out_NnzintileFile.write((char*)&max_nnz_intilev3, sizeof(int));
    out_NnzintileFile.close();
}
